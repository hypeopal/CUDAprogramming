#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#define CUDA_CHECK(statement) do {\
hipError_t code = statement;\
if (code != hipSuccess)\
{\
	printf("line: %d, error_code: %d, error_name: %s, error_description: %s\n",\
		__LINE__, code, hipGetErrorName(code), hipGetErrorString(code));\
	exit(EXIT_FAILURE); \
}\
} while(0) \

#define SIZE 512

__device__ int gpuSum = 0;

__global__ void sumArrayCUDA(const int* arr) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < SIZE)
		atomicAdd(&gpuSum, arr[tid]);
}

int sumArrayCPU(const int* arr) {
	int sum = 0;
	for (int i = 0; i < SIZE; ++i) sum += arr[i];
	return sum;
}


int main() {
	int* array = new int[SIZE];
	for (int i = 0; i < SIZE; ++i) {
		array[i] = i;
	}
	int cpuSum = sumArrayCPU(array);
	int size = sizeof(int) * SIZE;
	int* cudaArr;
	CUDA_CHECK(hipMalloc((void**)&cudaArr, size));
	CUDA_CHECK(hipMemcpy(cudaArr, array, size, hipMemcpyHostToDevice));
	int threadsPerBlock = 256;
	int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;
	sumArrayCUDA << <blocksPerGrid, threadsPerBlock >> > (cudaArr);
	CUDA_CHECK(hipDeviceSynchronize());

	int cudaSum;
	CUDA_CHECK(hipMemcpyFromSymbol(&cudaSum, HIP_SYMBOL(gpuSum), sizeof(int), 0, hipMemcpyDeviceToHost));

	std::cout << (cpuSum == cudaSum ? "correct" : "mistake") << std::endl;
	std::cout << cpuSum << "  " << cudaSum << std::endl;

	delete[] array;
	return 0;
}
