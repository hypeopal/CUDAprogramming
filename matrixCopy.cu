#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>
#include <numeric>

const int matrixSize = 1024;
const int tileSize = 32;
const int threadSize = 8;
const int testTimes = 10;
__constant__ int devMatrixSize;


__global__ void matrixCopyInRow(int* newMatrix, const int* oldMatrix) { //行优先
	int x = blockIdx.x * gridDim.x + threadIdx.x * 4;
	int y = blockIdx.y * gridDim.y + threadIdx.y;
	int i = 0;
	while (i < 4) {
		newMatrix[y * devMatrixSize + x + i] = oldMatrix[y * devMatrixSize + x + i];
		++i;
	}
}

__global__ void matrixCopyInCol(int* newMatrix, const int* oldMatrix) { //列优先
	int x = blockIdx.x * gridDim.x + threadIdx.x;
	int y = blockIdx.y * gridDim.y + threadIdx.y * 4;
	int i = 0;
	while (i < 4) {
		newMatrix[(y + i) * devMatrixSize + x] = oldMatrix[(y + i) * devMatrixSize + x];
		++i;
	}
}

hipError_t cuda_check(hipError_t error_code, int line)
{
	if (error_code != hipSuccess)
	{
		printf("line: %d, error_code: %d, error_name: %s, error_description: %s\n",
			line, error_code, hipGetErrorName(error_code), hipGetErrorString(error_code));
		exit(EXIT_FAILURE); // 检测到CUDA错误时退出程序  
	}
	return error_code;
}

void runMatrixCopy(const int* matrix, int* copiedMatrix, const std::string& type) {
	std::vector<float> runtimeList; //运行时间
	int size = sizeof(int) * matrixSize * matrixSize;
	int* cudaOldMatrix;
	cuda_check(hipMalloc((void**)&cudaOldMatrix, size), __LINE__);
	cuda_check(hipMemcpy(cudaOldMatrix, matrix, size, hipMemcpyHostToDevice), __LINE__);
	for (int n = 0; n < testTimes; ++n) {
		int* cudaNewMatrix;
		cuda_check(hipMalloc((void**)&cudaNewMatrix, size), __LINE__);
		
		dim3 dimGrid(tileSize, tileSize, 1);
		dim3 dimBlock;
		if (type == "Row") 
			dimBlock = dim3(threadSize, tileSize, 1);
		else 
			dimBlock = dim3(tileSize, threadSize, 1);

		// 记录GPU时间
		hipEvent_t start, stop;
		cuda_check(hipEventCreate(&start), __LINE__);
		cuda_check(hipEventCreate(&stop), __LINE__);

		//开始运行
		cuda_check(hipEventRecord(start, 0), __LINE__);
		if (type == "Row") 
			matrixCopyInRow <<<dimGrid, dimBlock >>> (cudaNewMatrix, cudaOldMatrix);
		else 
			matrixCopyInCol <<<dimGrid, dimBlock >>> (cudaNewMatrix, cudaOldMatrix);
		cuda_check(hipDeviceSynchronize(), __LINE__);
		cuda_check(hipEventRecord(stop, 0), __LINE__);

		cuda_check(hipEventSynchronize(stop), __LINE__);
		float gpuTime = 0;
		cuda_check(hipEventElapsedTime(&gpuTime, start, stop), __LINE__);
		runtimeList.emplace_back(gpuTime);

		cuda_check(hipMemcpy(copiedMatrix, cudaNewMatrix, size, hipMemcpyDeviceToHost), __LINE__);
		cuda_check(hipFree(cudaNewMatrix), __LINE__);

		//结果检查
		bool correct = true;
		for (int i = 0; i < matrixSize * matrixSize; ++i) {
			if (matrix[i] != copiedMatrix[i]) {
				correct = false;
				break;
			}
		}
		if (!correct) {
			std::cout << "Runtime error!" << std::endl;
			return;
		}
	}
	cuda_check(hipFree(cudaOldMatrix), __LINE__);
	std::cout << type << std::endl;
	std::cout << "Result:" << "correct" << std::endl;
	//计算平均时间
	std::cout << "Average runtime: " << std::accumulate(runtimeList.begin(), runtimeList.end(), 0.0f) / testTimes << std::endl;
}

int main() {
	int* matrix = new int[matrixSize * matrixSize]; //初始矩阵
	int* copiedMatrix = new int[matrixSize * matrixSize]; //复制后的矩阵
	//初始化矩阵
	for (int i = 0; i < matrixSize * matrixSize; ++i) {
		matrix[i] = i;
	}
	cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(devMatrixSize), &matrixSize, sizeof(int)), __LINE__);
	
	runMatrixCopy(matrix, copiedMatrix, "Row");
	std::cout << std::endl;
	runMatrixCopy(matrix, copiedMatrix, "Col");

	delete[] matrix;
	delete[] copiedMatrix;
	return 0;
}
