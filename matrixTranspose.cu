#include <iostream>
#include <hip/hip_runtime.h>


#define TILE_SIZE 32
#define MATRIX_SIZE 8192
const int testTimes = 10;


// GPU内核函数定义
__global__ void matrixCopyNaive(const int* matrix, int* copiedMatrix) {
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    for (int i = 0; i < TILE_SIZE; i += blockDim.y) {
        copiedMatrix[(y + i) * MATRIX_SIZE + x] = matrix[(y + i) * MATRIX_SIZE + x];
    }
}

__global__ void matrixCopyUsingSharedMem(const int* matrix, int* copiedMatrix) {
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    __shared__ int TILE[TILE_SIZE][TILE_SIZE];
    TILE[threadIdx.y][threadIdx.x] = matrix[(y) * MATRIX_SIZE + x];
    TILE[threadIdx.y + blockDim.y][threadIdx.x] = matrix[(y + blockDim.y)*MATRIX_SIZE + x];
    TILE[threadIdx.y + 2 * blockDim.y][threadIdx.x] = matrix[(y + 2 * blockDim.y) * MATRIX_SIZE + x];
    TILE[threadIdx.y + 3 * blockDim.y][threadIdx.x] = matrix[(y + 3 * blockDim.y) * MATRIX_SIZE + x];
    
    copiedMatrix[(y)*MATRIX_SIZE + x] = TILE[threadIdx.y][threadIdx.x];
    copiedMatrix[(y + blockDim.y) * MATRIX_SIZE + x] = TILE[threadIdx.y + blockDim.y][threadIdx.x];
    copiedMatrix[(y + 2 * blockDim.y) * MATRIX_SIZE + x] = TILE[threadIdx.y + 2 * blockDim.y][threadIdx.x];
    copiedMatrix[(y + 3 * blockDim.y) * MATRIX_SIZE + x] = TILE[threadIdx.y + 3 * blockDim.y][threadIdx.x];
}

__global__ void matrixTransposeNaive(const int* matrix, int* transposedMatrix) {
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    for (int i = 0; i < TILE_SIZE; i += blockDim.y) {
        transposedMatrix[x * MATRIX_SIZE + (y + i)] = matrix[(y + i) * MATRIX_SIZE + x];
    }
}

__global__ void matrixTransposeUsingSharedMem(const int* matrix, int* transposedMatrix) {
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;
    __shared__ int Tile[TILE_SIZE][TILE_SIZE];
    for (int i = 0; i < TILE_SIZE; i += blockDim.y) {
        Tile[threadIdx.y + i][threadIdx.x] = matrix[(y + i) * MATRIX_SIZE + x];
    }
    __syncthreads();
    x = blockIdx.y * TILE_SIZE + threadIdx.x;  // 块内转置
    y = blockIdx.x * TILE_SIZE + threadIdx.y;
    for (int i = 0; i < TILE_SIZE; i += blockDim.y) {
        transposedMatrix[(y + i) * MATRIX_SIZE + x] = Tile[threadIdx.x][threadIdx.y + i];
    }
}

__global__ void matrixTransposeUsingSharedMemWithPadding(const int* matrix, int* transposedMatrix) {
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    __shared__ int Tile[(TILE_SIZE + 1) * TILE_SIZE];
    for (int i = 0; i < TILE_SIZE; i += blockDim.y) {
        Tile[(threadIdx.y + i) * (TILE_SIZE + 1) + threadIdx.x] = matrix[(y + i) * MATRIX_SIZE + x];
    }

    __syncthreads();
    x = blockIdx.y * TILE_SIZE + threadIdx.x;  // 块内转置
    y = blockIdx.x * TILE_SIZE + threadIdx.y;
    for (int i = 0; i < TILE_SIZE; i += blockDim.y) {
        transposedMatrix[(y + i) * MATRIX_SIZE + x] = Tile[threadIdx.x * (TILE_SIZE + 1) + (threadIdx.y + i)];
    }
}

hipError_t cuda_check(const hipError_t& error_code, int line) {
    if (error_code != hipSuccess)
    {
        printf("line: %d, error_code: %d, error_name: %s, error_description: %s\n",
            line, error_code, hipGetErrorName(error_code), hipGetErrorString(error_code));
        exit(EXIT_FAILURE); // 检测到CUDA错误时退出程序
    }
    return error_code;
}

// CPU矩阵运算
void matrixTransposeByCPU(const int* matrix, int* transposedMatrix) {
    for (int i = 0; i < MATRIX_SIZE; ++i)
        for (int k = 0; k < MATRIX_SIZE; ++k)
            transposedMatrix[k * MATRIX_SIZE + i] = matrix[i * MATRIX_SIZE + k];
}

void matrixCopyByCPU(const int* matrix, int* copiedMatrix) {
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i)
        copiedMatrix[i] = matrix[i];
}

bool checkResult(const int* matrix1, const int* matrix2) {
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i)
        if (matrix1[i] != matrix2[i]) return false;
    return true;
}

// CUDA核函数模板
template <typename Kernel>
void runCudaKernel(const int* matrix, int* resultMatrix, Kernel kernel, const char* label) {
    int size = sizeof(int) * MATRIX_SIZE * MATRIX_SIZE;
    int* cudaOldMatrix;
    int* cudaResultMatrix;
    // 分配GPU内存
    cuda_check(hipMalloc((void**)&cudaOldMatrix, size), __LINE__);
    cuda_check(hipMemcpy(cudaOldMatrix, matrix, size, hipMemcpyHostToDevice), __LINE__);
    cuda_check(hipMalloc((void**)&cudaResultMatrix, size), __LINE__);

    // 定义线程块和网格
    dim3 dimGrid(MATRIX_SIZE / TILE_SIZE, MATRIX_SIZE / TILE_SIZE, 1);
    dim3 dimBlock(TILE_SIZE, 8, 1);

    // 创建CUDA计时事件
    hipEvent_t start, stop;
    cuda_check(hipEventCreate(&start), __LINE__);
    cuda_check(hipEventCreate(&stop), __LINE__);
    float gpuTime = 0;

    // 启动核函数并测量执行时间
    cuda_check(hipEventRecord(start, 0), __LINE__);
    for (int i = 0; i < testTimes; ++i) {
        kernel <<<dimGrid, dimBlock >>> (cudaOldMatrix, cudaResultMatrix);
        cuda_check(hipDeviceSynchronize(), __LINE__);
    }
    cuda_check(hipEventRecord(stop, 0), __LINE__);
    cuda_check(hipEventSynchronize(stop), __LINE__);
    cuda_check(hipEventElapsedTime(&gpuTime, start, stop), __LINE__);

    std::cout << label << gpuTime / testTimes << " ms" << std::endl;

    // 将结果复制回主机并释放GPU内存
    cuda_check(hipMemcpy(resultMatrix, cudaResultMatrix, size, hipMemcpyDeviceToHost), __LINE__);
    cuda_check(hipFree(cudaResultMatrix), __LINE__);
    cuda_check(hipFree(cudaOldMatrix), __LINE__);
}

int main() {
    int* matrix = new int[MATRIX_SIZE * MATRIX_SIZE];  // 原矩阵
    int* transposedMatrix = new int[MATRIX_SIZE * MATRIX_SIZE];  // cpu转置后的矩阵
    int* devTransposedMatrix = new int[MATRIX_SIZE * MATRIX_SIZE];  // gpu转置的矩阵
    int* copiedMatrix = new int[MATRIX_SIZE * MATRIX_SIZE];  // cpu复制的矩阵
    int* devCopiedMatrix = new int[MATRIX_SIZE * MATRIX_SIZE];  // gpu复制的矩阵

    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i) { // 初始化矩阵
        matrix[i] = i - (MATRIX_SIZE * MATRIX_SIZE / 2);
    }

    // 矩阵转置
    matrixTransposeByCPU(matrix, transposedMatrix);
    runCudaKernel(matrix, devTransposedMatrix, matrixTransposeNaive, 
        "CUDA matrix transpose naive average runtime:");
    std::cout << "Transpose Result: " << (checkResult(transposedMatrix, devTransposedMatrix) ? "correct" : "error") << std::endl;
    runCudaKernel(matrix, devTransposedMatrix, matrixTransposeUsingSharedMem, 
        "CUDA matrix transpose with shared memory average runtime:");
    std::cout << "Transpose Result: " << (checkResult(transposedMatrix, devTransposedMatrix) ? "correct" : "error") << std::endl;
    runCudaKernel(matrix, devTransposedMatrix, matrixTransposeUsingSharedMemWithPadding,
        "CUDA matrix transpose with shared memory and padding average runtime:");
    std::cout << "Transpose Result: " << (checkResult(transposedMatrix, devTransposedMatrix) ? "correct" : "error") << std::endl;
    
    std::cout << std::endl;

    // 矩阵复制
    matrixCopyByCPU(matrix, copiedMatrix);
    runCudaKernel(matrix, devCopiedMatrix, matrixCopyNaive, 
        "CUDA matrix copy naive average runtime:");
    std::cout << "Copy Result: " << (checkResult(copiedMatrix, devCopiedMatrix) ? "correct" : "error") << std::endl;
    runCudaKernel(matrix, devCopiedMatrix, matrixCopyUsingSharedMem, 
        "CUDA matrix copy with shared memory average runtime:");
    std::cout << "Copy Result: " << (checkResult(copiedMatrix, devCopiedMatrix) ? "correct" : "error") << std::endl;

    // 释放内存
    delete[] matrix;
    delete[] transposedMatrix;
    delete[] devTransposedMatrix;
    delete[] copiedMatrix;
    delete[] devCopiedMatrix;
    return 0;
}
