#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#define TILE_SIZE 32
const int matrixSize = 1024;
const int tileSize = 32;
const int threadSize = 8;
const int testTimes = 10;
__constant__ int devMatrixSize;

__global__ void matrixCopyInCol(int* newMatrix, const int* oldMatrix, int numPerThread) { //列优先
	int x = blockIdx.x * gridDim.x + threadIdx.x;
	int y = blockIdx.y * gridDim.y + threadIdx.y * numPerThread;
	int i = 0;
	while (i < numPerThread) {
		newMatrix[(y + i) * devMatrixSize + x] = oldMatrix[(y + i) * devMatrixSize + x];
		++i;
	}
}

__global__ void matrixTransposeUsingSharedMem(const int* matrix, int* transposedMatrix) {
	int x = blockIdx.x * gridDim.x + threadIdx.x;
	int y = blockIdx.y * gridDim.y + threadIdx.y;

	__shared__ int Tile[TILE_SIZE][TILE_SIZE];
	for (int i = 0; i < TILE_SIZE; ++i) {
		Tile[threadIdx.x][threadIdx.y + i] = matrix[(y + i) * devMatrixSize + x];
	}

	__syncthreads();

	for (int i = 0; i < TILE_SIZE; ++i) {
		transposedMatrix[x * devMatrixSize + (y + i)] = Tile[threadIdx.x][threadIdx.y + i];
	}
}

hipError_t cuda_check(const hipError_t& error_code, int line)
{
	if (error_code != hipSuccess)
	{
		printf("line: %d, error_code: %d, error_name: %s, error_description: %s\n",
			line, error_code, hipGetErrorName(error_code), hipGetErrorString(error_code));
		exit(EXIT_FAILURE); // 检测到CUDA错误时退出程序
	}
	return error_code;
}

void matrixTransposeByCPU(const int* matrix, int* transposedMatrix) {
	for (int i = 0; i < matrixSize; ++i) {
		for (int k = 0; k < matrixSize; ++k) {
			transposedMatrix[k * matrixSize + i] = matrix[i * matrixSize + k];
		}
	}
}

bool checkResult(const int* matrix, const int* transposedMatrix) { //转置正确性检测
	for (int i = 0; i < matrixSize; ++i) {
		for (int k = 0; k < matrixSize; ++k) {
			if (matrix[i * matrixSize + k] != transposedMatrix[i * matrixSize + k])
				return false;
		}
	}
	return true;
}

void runCudaMatrixTranspose(const int* matrix, int* transposedMatrix) {
	//GPU内存分配
	int size = sizeof(int) * matrixSize * matrixSize;
	int* cudaOldMatrix;
	int* cudaTransposedMatrix;
	cuda_check(hipMalloc((void**)&cudaOldMatrix, size), __LINE__);
	cuda_check(hipMemcpy(cudaOldMatrix, matrix, size, hipMemcpyHostToDevice), __LINE__);
	cuda_check(hipMalloc((void**)&cudaTransposedMatrix, size), __LINE__);
	//GPU常量分配
	cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(devMatrixSize), &matrixSize, sizeof(int)), __LINE__);

	dim3 dimGrid(tileSize, tileSize, 1);
	dim3 dimBlock(tileSize, 1, 1);

	// 记录GPU时间
	hipEvent_t start, stop;
	cuda_check(hipEventCreate(&start), __LINE__);
	cuda_check(hipEventCreate(&stop), __LINE__);

	cuda_check(hipEventRecord(start, 0), __LINE__);
	matrixTransposeUsingSharedMem << <dimGrid, dimBlock >> > (cudaOldMatrix, cudaTransposedMatrix);
	cuda_check(hipDeviceSynchronize(), __LINE__);
	cuda_check(hipEventRecord(stop, 0), __LINE__);

	cuda_check(hipEventSynchronize(stop), __LINE__);
	float gpuTime = 0;
	cuda_check(hipEventElapsedTime(&gpuTime, start, stop), __LINE__);
	std::cout << "CUDA matrix transpose with shared memory runtime: " << gpuTime << std::endl;

	cuda_check(hipMemcpy(transposedMatrix, cudaTransposedMatrix, size, hipMemcpyDeviceToHost), __LINE__);
	cuda_check(hipFree(cudaTransposedMatrix), __LINE__);
	cuda_check(hipFree(cudaOldMatrix), __LINE__);
}

int main() {
	int* matrix = new int[matrixSize * matrixSize]; //初始矩阵
	int* transposedMatrix = new int[matrixSize * matrixSize]; //CPU转置的矩阵
	int* devTransposedMatrix = new int[matrixSize * matrixSize]; //GPU转置的矩阵

	//初始化矩阵
	for (int i = 0; i < matrixSize * matrixSize; ++i) {
		matrix[i] = i;
	}
	
	runCudaMatrixTranspose(matrix, devTransposedMatrix); //CUDA转置
	matrixTransposeByCPU(matrix, transposedMatrix); //CPU转置
	
	std::cout << "Result: " << (checkResult(transposedMatrix, devTransposedMatrix) ? "correct" : "error") << std::endl;

	delete[] matrix;
	delete[] transposedMatrix;
	delete[] devTransposedMatrix;
	return 0;
}
